#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define N 4096
#define COLS 256

/* Euklidska dist. (L2 norm)*/
__global__ void l2norm_kernel(float* A, float* out, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float sum = 0.0f;
        for (int i = 0; i < cols; i++) {
            float val = A[idx * cols + i];
            sum += val * val;
        }
        out[idx] = sqrtf(sum);
    }
}

int main() {
    float* h_A = (float*)malloc(N * COLS * sizeof(float));
    float* h_out = (float*)malloc(N * sizeof(float));
    float* d_A, * d_out;

    for (int i = 0; i < N * COLS; i++)
        h_A[i] = rand() % 10;

    hipMalloc(&d_A, N * COLS * sizeof(float));
    hipMalloc(&d_out, N * sizeof(float));
    hipMemcpy(d_A, h_A, N * COLS * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockDim(256);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    l2norm_kernel << <gridDim, blockDim >> > (d_A, d_out, COLS);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time_ms = 0;
    hipEventElapsedTime(&time_ms, start, stop);

    hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("CUDA L2 norm kernel time: %f ms \n", time_ms);
    printf("Last few results: \n");
    for (int i = N - 5; i < N; i++) {
        printf("%f\\n", h_out[i]);
    }

    hipFree(d_A);
    hipFree(d_out);
    free(h_A);
    free(h_out);

    return 0;
}
