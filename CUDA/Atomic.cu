#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>


__global__ void vectSumRace(int* d_vect, size_t size, int* result) {
	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	while (tid < size) {
		*result += d_vect[tid];
		tid += blockDim.x * gridDim.x;
	}
}

__global__ void vectSumAtomic(int* d_vect, size_t size, int* result) {
	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	while (tid < size) {
        atomicAdd(result, d_vect[tid]);
		tid += blockDim.x * gridDim.x;
	}
}



int main()
{
    int vect[1000] = { 0 };
    int result=0, result2=0;

    for (int i = 0; i < 1000; i++) {
        vect[i] = i;
    }


    int* dev_vect = 0;
    int* dev_res = 0;
    int* dev_res2 = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_vect, 1000 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_res, 1 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_res2, 1 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_vect, vect, 1000 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_res, &result, 1 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_res2, &result2, 1 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    vectSumRace <<<1, 1000 >>> (dev_vect, 1000, dev_res);
    vectSumAtomic << <1, 1000 >> > (dev_vect, 1000, dev_res2);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(&result, dev_res, 1 * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    std::cout << "Sum-race: "<<result << "\n\n";


    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(&result2, dev_res2, 1 * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

   
    std::cout << "Sum-atomic: "<<result2;
    

Error:
    hipFree(dev_res);
    hipFree(dev_vect);
 



    return 0;
}